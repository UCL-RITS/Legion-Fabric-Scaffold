
#include <hip/hip_runtime.h>
#include <stdio.h>

void handle_error( hipError_t error, const char* message)
{
	if(error!=hipSuccess) {
		fprintf(stderr,"ERROR: %s : %s\n",message,hipGetErrorString(error));
		exit(-1);
	}
}
