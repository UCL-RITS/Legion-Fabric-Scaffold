
#include <hip/hip_runtime.h>
#include <stdio.h>

void handle_error( hipError_t error, const char* message)
{
	if(error!=hipSuccess) {
		fprintf(stderr,"ERROR: %s : %s\n",message,hipGetErrorString(error));
		exit(-1);
	}
}

__global__ void reduce_add (int * array, int * result){

  // Here's how to do an O(ln N) reduce in pure
  // low-level CUDA.

	extern __shared__ int cache[];

	int thread_id=threadIdx.x;

	cache[thread_id]=array[thread_id];
	int reduction_index=blockDim.x/2;

  __syncthreads();

  while (reduction_index!=0){
		if (thread_id<reduction_index){
			cache[thread_id]+=cache[thread_id+reduction_index];
		}
		__syncthreads();
		reduction_index/=2;
	}

  if (thread_id==0) {
		*result=cache[0];
	}
}
