#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/device_vector.h>

#include "helpers.h"
#include "Model.h"


__global__ void map (int * results);

const int thread_count=32;

int main(int argc, char **argv){


	dim3 tpb(thread_count,1,1);

  int host_total=0;

  thrust::device_vector<int> results(thread_count);
  thrust::device_vector<int> device_total(1);

  map <<< 1, tpb >>> (thrust::raw_pointer_cast(&results[0]));

  reduce_add <<< 1, tpb, tpb.x >>> (
          thrust::raw_pointer_cast(&results[0]),
          thrust::raw_pointer_cast(&device_total[0]));

  host_total = device_total[0];

  printf("Final Result: %i\n", host_total);

}

__global__ void map (int * results){
    Model *m= new Model(threadIdx.x, blockDim.x);
    results[threadIdx.x] = m->calculate();
}
