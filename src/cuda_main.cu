#include "hip/hip_runtime.h"
#include <stdio.h>
#include "helpers.h"
#include "Model.h"


__global__ void map (int * results);

const int thread_count=32;

int main(int argc, char **argv){


	dim3 tpb(thread_count,1,1);

  int* results=0;
  int* device_total=0;
  int host_total=0;

  handle_error(hipMalloc( &results,
      thread_count*sizeof( int ) ), "Allocate device samples");

  handle_error(hipMalloc( &device_total,
      sizeof(int)), "Allocate result");

  map <<< 1, tpb >>> (results);
  reduce_add <<< 1, tpb, tpb.x >>> (results, device_total);

  handle_error(hipMemcpy(&host_total,
      device_total, sizeof(int), hipMemcpyDeviceToHost), "Retrieve result");

  printf("Final Result: %i\n", host_total);

}

__global__ void map (int * results){
    Model *m= new Model(threadIdx.x, blockDim.x);
    results[threadIdx.x] = m->calculate();
}
