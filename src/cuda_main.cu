#include "hip/hip_runtime.h"
#include <stdio.h>
#include "helpers.h"

__global__ void map (int * results);
__global__ void reduce (int * array, int * result);


const int thread_count=32;

int main(int argc, char **argv){


	dim3 tpb(thread_count,1,1);

  int* results=0;
  int* device_total=0;
  int host_total=0;

  handle_error(hipMalloc( &results,
      thread_count*sizeof( int ) ), "Allocate device samples");

  handle_error(hipMalloc( &device_total,
      sizeof(int)), "Allocate result");

  map <<< 1, tpb >>> (results);
  reduce <<< 1, tpb >>> (results, device_total);

  handle_error(hipMemcpy(&host_total,
      device_total, sizeof(int),hipMemcpyDeviceToHost), "Retrieve result");

  printf("Final Result: %i\n", host_total);

}

__global__ void map (int * results){
    results[threadIdx.x] = threadIdx.x;
}

__global__ void reduce (int * array, int * result){

  // Here's how to do an O(ln N) reduce in pure
  // low-level CUDA.

	__shared__ int cache[thread_count];
	int thread_id=threadIdx.x;

	cache[thread_id]=array[thread_id];
	int reduction_index=thread_count/2;

  __syncthreads();

  while (reduction_index!=0){
		if (thread_id<reduction_index){
			cache[thread_id]+=cache[thread_id+reduction_index];
		}
		__syncthreads();
		reduction_index/=2;
	}

  if (thread_id==0) {
		*result=cache[0];
	}
}
