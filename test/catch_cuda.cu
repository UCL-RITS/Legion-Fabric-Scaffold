#include "hip/hip_runtime.h"
// Next line tells CATCH we will use our own main function
#define CATCH_CONFIG_RUNNER
#include <cmath>
#include <thrust/device_vector.h>
#include "catch.hpp"
#include "Model.h"
#include "helpers.h"


int main(int argc, char * argv[]) {
    int result = Catch::Session().run(argc, argv);
    return result;
}

__global__ void threadnumber (int * results){
    results[threadIdx.x] = threadIdx.x;
}


TEST_CASE ("CUDA Tests"){

    SECTION("Basic cuda dispatch works"){


      const int thread_count=32;
      dim3 tpb(thread_count,1,1);

      int* device_results=0;
      int host_results[thread_count];

      handle_error(hipMalloc( &device_results,
            thread_count*sizeof( int ) ), "Allocate device samples");

      threadnumber <<< 1, tpb >>> (device_results);

      handle_error(hipMemcpy(&host_results,
          device_results, sizeof(int)*thread_count,
          hipMemcpyDeviceToHost), "Retrieve result");

      for (unsigned int i=0; i<thread_count; i++) {
        REQUIRE(host_results[i]==i);
      }
    }

    SECTION("Thrust pointer cuda dispatch works"){


      const int thread_count=32;
      dim3 tpb(thread_count,1,1);

      thrust::device_vector<int> device_results(thread_count);
      thrust::host_vector<int> host_results(thread_count);

      threadnumber <<< 1, tpb >>> (thrust::raw_pointer_cast(&device_results[0]));

      host_results=device_results;

      for (unsigned int i=0; i<thread_count; i++) {
        REQUIRE(host_results[i]==i);
      }
    }

    SECTION("Thrust dispatch works"){

      const int thread_count=32;
      thrust::device_vector<int> device_results(thread_count);

    	thrust::generate(device_results.begin(), device_results.end(),
    		[]__device__() {
    			return threadIdx.x;
    		} );

      thrust::host_vector<int> host_results(thread_count);

      host_results=device_results;

      for (unsigned int i=0; i<thread_count; i++) {
        REQUIRE(host_results[i]==i);
      }
    }
}
